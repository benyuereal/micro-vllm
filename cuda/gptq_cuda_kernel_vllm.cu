#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <hipblas.h>
#include <ATen/cuda/HIPContext.h>

// vLLM风格的分块大小
#define BLOCK_KN_SIZE 128
#define BLOCK_M_SIZE_MAX 8
#define THREADS_X 32
#define THREADS_Y 32
#define DIVIDE(x, size) (((x) + (size) - 1) / (size))

// vLLM风格的向量化点积
__forceinline__ __device__ half2 dot22_8(half2 (&dq)[4], const half* a_ptr,
                                         const half2 g_result) {
  half2 result = {};
  const half2* a2_ptr = (const half2*)a_ptr;
#pragma unroll
  for (int i = 0; i < 4; i++) result = __hfma2(dq[i], *a2_ptr++, result);
  return __hadd2(result, g_result);
}

__forceinline__ __device__ float dot22_8_f(half2 (&dq)[4], const half* a_ptr,
                                           const float g_result,
                                           const float qs_f) {
  half2 result = {};
  const half2* a2_ptr = (const half2*)a_ptr;
#pragma unroll
  for (int i = 0; i < 4; i++) result = __hfma2(dq[i], *a2_ptr++, result);
  float result_f =
      __half2float(__low2half(result)) + __half2float(__high2half(result));
  return fma(result_f, qs_f, g_result);
}

// vLLM风格的4bit反量化
__forceinline__ __device__ void dequant_4bit_8_gptq(
    uint32_t qw, half2 (&dq)[4], const half2 (&z1z16)[2], 
    const half2 (&y1y16)[2], int size_n, bool alt) {
  // 提取4bit值并反量化
  for (int i = 0; i < 4; i++) {
    int w = (qw >> (i * 8)) & 0xFF;
    half2 w01 = __halves2half2(__int2half_rn(w & 0xF), __int2half_rn(w >> 4));
    dq[i] = __hfma2(w01, y1y16[i % 2], z1z16[i % 2]);
  }
}

// vLLM风格的主内核
template <bool first_block, int m_count>
__global__ void gemm_half_q_half_gptq_4bit_kernel(
    const half* __restrict__ a, const uint32_t* __restrict__ b_q_weight,
    const uint32_t* __restrict__ b_gptq_qzeros,
    const half* __restrict__ b_gptq_scales, half* __restrict__ c,
    const int size_m, const int size_n, const int size_k, const int groups,
    const int* __restrict__ b_q_perm) {
  
  auto t = threadIdx.x;
  
  // Block
  auto offset_n = blockIdx.x * BLOCK_KN_SIZE * 4;
  auto offset_m = blockIdx.y * m_count;
  auto offset_k = blockIdx.z * BLOCK_KN_SIZE;
  
  int end_k = min(offset_k + BLOCK_KN_SIZE, size_k);
  
  int n = offset_n + t * 4;
  
  // Preload block_a
  __shared__ half block_a[m_count][BLOCK_KN_SIZE];
  
  if (offset_k + t < end_k) {
    for (int m = 0; m < m_count; ++m) {
      const half* a_ptr = a + (offset_m + m) * size_k;
      half* block_a_ptr = block_a[m];
      
      half a0;
      if (b_q_perm)
        a0 = a_ptr[b_q_perm[offset_k + t]];
      else
        a0 = a_ptr[offset_k + t];
      block_a_ptr[t] = a0;
    }
  }
  
  // Zero output
  if (n >= size_n) return;
  
  if (blockIdx.z == 0) {
    for (int m = 0; m < m_count; m++)
      *((uint64_t*)(c + (offset_m + m) * size_n + n)) = 0;
  }
  
  __syncthreads();
  
  // Find initial group
  int groupsize = size_k / groups;
  int group = offset_k / groupsize;
  int nextgroup = offset_k + groupsize;
  
  // a, b offset
  int qk = offset_k / (32 / 4);
  
  const uint32_t* b_ptr = b_q_weight + qk * size_n + n;
  const half* a_ptr = &block_a[0][0];
  int a_stride = BLOCK_KN_SIZE;
  
  // Initial group
  int zeros[4];
  float scales[4];
  half2 z1z16[4][2];
  half2 y1y16[4][2];
  
  // 加载zeros和scales
  for (int i = 0; i < 4; i++) {
    zeros[i] = b_gptq_qzeros[group * size_n + n + i] + 1;
    scales[i] = __half2float(b_gptq_scales[group * size_n + n + i]);
  }
  
  // Column result
  float block_c[m_count][4] = {};
  
  // Dequantize and multiply
  int k = offset_k;
  while (k < end_k) {
    if (k == nextgroup) {
      group++;
      nextgroup += groupsize;
      for (int i = 0; i < 4; i++) {
        zeros[i] = b_gptq_qzeros[group * size_n + n + i] + 1;
        scales[i] = __half2float(b_gptq_scales[group * size_n + n + i]);
      }
    }
    
#pragma unroll
    for (int j = 0; j < 4; j++) {
      const int4* b_ptr4 = (int4*)b_ptr;
      int4 load_int4 = *b_ptr4;
      
      half2 dq[4][4];
      dequant_4bit_8_gptq(load_int4.x, dq[0], z1z16[0], y1y16[0], size_n, false);
      dequant_4bit_8_gptq(load_int4.y, dq[1], z1z16[1], y1y16[1], size_n, false);
      dequant_4bit_8_gptq(load_int4.z, dq[2], z1z16[2], y1y16[2], size_n, false);
      dequant_4bit_8_gptq(load_int4.w, dq[3], z1z16[3], y1y16[3], size_n, false);
      
#pragma unroll
      for (int m = 0; m < m_count; m++) {
        block_c[m][0] = fma(dot22_8_f(dq[0], a_ptr + m * a_stride), scales[0], block_c[m][0]);
        block_c[m][1] = fma(dot22_8_f(dq[1], a_ptr + m * a_stride), scales[1], block_c[m][1]);
        block_c[m][2] = fma(dot22_8_f(dq[2], a_ptr + m * a_stride), scales[2], block_c[m][2]);
        block_c[m][3] = fma(dot22_8_f(dq[3], a_ptr + m * a_stride), scales[3], block_c[m][3]);
      }
      
      b_ptr += size_n;
      a_ptr += 8;
    }
    
    k += 32;
  }
  
  for (int m = 0; m < m_count; m++) {
    half2* out = (half2*)(c + (offset_m + m) * size_n + n);
    half2 result01 = __halves2half2(__float2half_rn(block_c[m][0]), __float2half_rn(block_c[m][1]));
    half2 result23 = __halves2half2(__float2half_rn(block_c[m][2]), __float2half_rn(block_c[m][3]));
    atomicAdd(out, result01);
    atomicAdd(out + 1, result23);
  }
}

// 内核选择函数
typedef void (*fp_gemm_half_q_half_gptq_kernel)(const half*, const uint32_t*,
                                                const uint32_t*, const half*,
                                                half*, const int, const int,
                                                const int, const int,
                                                const int*);

template <bool first_block, int m_count>
__global__ void gemm_half_q_half_gptq_4bit_kernel_wrapper(
    const half* __restrict__ a, const uint32_t* __restrict__ b_q_weight,
    const uint32_t* __restrict__ b_gptq_qzeros,
    const half* __restrict__ b_gptq_scales, half* __restrict__ c,
    const int size_m, const int size_n, const int size_k, const int groups,
    const int* __restrict__ b_q_perm) {
  gemm_half_q_half_gptq_4bit_kernel<first_block, m_count><<<1, 1>>>(
      a, b_q_weight, b_gptq_qzeros, b_gptq_scales, c, size_m, size_n, size_k, groups, b_q_perm);
}

fp_gemm_half_q_half_gptq_kernel pick_gemm_half_q_half_gptq_kernel(
    bool first_block, const int m_count, const int bit) {
  if (m_count == 1) {
    if (bit == 4) return gemm_half_q_half_gptq_4bit_kernel_wrapper<true, 1>;
  }
  return NULL;
}

// cuBLAS辅助函数
hipblasHandle_t get_cublas_handle() {
  static hipblasHandle_t handle = nullptr;
  if (handle == nullptr) {
    hipblasCreate(&handle);
  }
  return handle;
}

// Python接口
torch::Tensor fused_gptq_gemm_4bit_cuda(
    torch::Tensor input,
    torch::Tensor qweight,
    torch::Tensor qzeros,
    torch::Tensor scales,
    int groupsize
) {
  // 获取维度
  int M = input.size(0);
  int K = input.size(1);
  int N = qweight.size(0);
  
  // 创建输出张量
  auto output = torch::zeros({M, N}, torch::TensorOptions()
      .dtype(input.dtype())
      .device(input.device()));
  
  // vLLM风格的网格和块大小
  dim3 blockDim, gridDim;
  blockDim.x = BLOCK_KN_SIZE;
  blockDim.y = 1;
  blockDim.z = 1;
  gridDim.x = DIVIDE(N, BLOCK_KN_SIZE * 4);
  gridDim.y = DIVIDE(M, BLOCK_M_SIZE_MAX);
  gridDim.z = DIVIDE(K, BLOCK_KN_SIZE);
  
  // 选择内核
  fp_gemm_half_q_half_gptq_kernel kernel = pick_gemm_half_q_half_gptq_kernel(true, 1, 4);
  
  if (kernel) {
    // 使用vLLM风格内核
    const hipStream_t stream = c10::cuda::getCurrentCUDAStream();
    kernel<<<gridDim, blockDim, 0, stream>>>(
        reinterpret_cast<const half*>(input.data_ptr<at::Half>()),
        qweight.data_ptr<uint32_t>(),
        qzeros.data_ptr<uint32_t>(),
        reinterpret_cast<const half*>(scales.data_ptr<at::Half>()),
        reinterpret_cast<half*>(output.data_ptr<at::Half>()),
        M, N, K, K / groupsize, nullptr
    );
  } else {
    // 回退到cuBLAS
    hipblasHandle_t handle = get_cublas_handle();
    const half alpha = __float2half(1.0f);
    const half beta = __float2half(0.0f);
    
    // 简化的cuBLAS调用
    hipblasHgemm(
        handle,
        HIPBLAS_OP_N, HIPBLAS_OP_N,
        N, M, K,
        &alpha,
        reinterpret_cast<const half*>(input.data_ptr<at::Half>()), K,
        reinterpret_cast<const half*>(input.data_ptr<at::Half>()), K,
        &beta,
        reinterpret_cast<half*>(output.data_ptr<at::Half>()), N
    );
  }
  
  hipDeviceSynchronize();
  return output;
}

// PyTorch绑定
PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("fused_gptq_gemm_4bit_cuda", &fused_gptq_gemm_4bit_cuda, "vLLM Style Fused GPTQ GEMM 4bit CUDA");
}

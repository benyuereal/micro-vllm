#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <hip/hip_fp16.h>

// vLLM风格的分块大小
#define BLOCK_KN_SIZE 128
#define BLOCK_M_SIZE_MAX 8
#define THREADS_X 32
#define THREADS_Y 32
#define DIVIDE(x, size) (((x) + (size) - 1) / (size))

// vLLM风格的向量化点积
__forceinline__ __device__ half2 dot22_8(half2 (&dq)[4], const half* a_ptr,
                                         const half2 g_result) {
  half2 result = {};
  const half2* a2_ptr = (const half2*)a_ptr;
#pragma unroll
  for (int i = 0; i < 4; i++) result = __hfma2(dq[i], *a2_ptr++, result);
  return __hadd2(result, g_result);
}

__forceinline__ __device__ float dot22_8_f(half2 (&dq)[4], const half* a_ptr,
                                           const float g_result,
                                           const float qs_f) {
  half2 result = {};
  const half2* a2_ptr = (const half2*)a_ptr;
#pragma unroll
  for (int i = 0; i < 4; i++) result = __hfma2(dq[i], *a2_ptr++, result);
  float result_f =
      __half2float(__low2half(result)) + __half2float(__high2half(result));
  return fma(result_f, qs_f, g_result);
}

// vLLM风格的4bit反量化
__forceinline__ __device__ void dequant_4bit_8_gptq(
    uint32_t qw, half2 (&dq)[4], const half2 (&z1z16)[2], 
    const half2 (&y1y16)[2], int size_n, bool alt) {
  // 提取4bit值并反量化
  for (int i = 0; i < 4; i++) {
    int w = (qw >> (i * 8)) & 0xFF;
    half2 w01 = __halves2half2(__int2half_rn(w & 0xF), __int2half_rn(w >> 4));
    dq[i] = __hfma2(w01, y1y16[i % 2], z1z16[i % 2]);
  }
}

// vLLM风格的主内核
template <bool first_block, int m_count>
__global__ void gemm_half_q_half_gptq_4bit_kernel(
    const half* __restrict__ a, const uint32_t* __restrict__ b_q_weight,
    const uint32_t* __restrict__ b_gptq_qzeros,
    const half* __restrict__ b_gptq_scales, half* __restrict__ c,
    const int size_m, const int size_n, const int size_k, const int groups) {
  
  auto t = threadIdx.x;
  
  // 分块
  auto offset_n = blockIdx.x * BLOCK_KN_SIZE * 4;
  auto offset_m = blockIdx.y * m_count;
  auto offset_k = blockIdx.z * BLOCK_KN_SIZE;
  
  int end_k = min(offset_k + BLOCK_KN_SIZE, size_k);
  int n = offset_n + t * 4;
  
  // 预加载block_a到共享内存
  __shared__ half block_a[m_count][BLOCK_KN_SIZE];
  
  if (offset_k + t < end_k) {
    for (int m = 0; m < m_count; ++m) {
      block_a[m][t] = a[(offset_m + m) * size_k + offset_k + t];
    }
  }
  
  // 零化输出
  if (n >= size_n) return;
  
  if (blockIdx.z == 0) {
    for (int m = 0; m < m_count; m++)
      *((uint64_t*)&c[(offset_m + m) * size_n + n]) = 0;
  }
  
  __syncthreads();
  
  // 找到初始组
  int groupsize = size_k / groups;
  int group = offset_k / groupsize;
  int nextgroup = offset_k + groupsize;
  
  // 计算偏移
  int qk = offset_k / (32 / 4);
  const uint32_t* b_ptr = b_q_weight + qk * size_n + n;
  const half* a_ptr = &block_a[0][0];
  int a_stride = BLOCK_KN_SIZE;
  
  // 初始组
  int zeros[4];
  float scales[4];
  half2 z1z16[4][2];
  half2 y1y16[4][2];
  
  // 加载零点和缩放因子
  for (int i = 0; i < 4; i++) {
    zeros[i] = b_gptq_qzeros[group * (size_n / 8) + (n + i) / 8];
    scales[i] = __half2float(b_gptq_scales[group * size_n + n + i]);
  }
  
  // 预计算反量化参数
  for (int i = 0; i < 4; i++) {
    int zero_val = (zeros[i] >> ((n + i) % 8 * 4)) & 0xF;
    z1z16[i][0] = __halves2half2(__int2half_rn(-zero_val - 1), __int2half_rn(-zero_val - 1));
    z1z16[i][1] = __halves2half2(__int2half_rn(-zero_val - 1), __int2half_rn(-zero_val - 1));
    y1y16[i][0] = __halves2half2(__float2half(scales[i]), __float2half(scales[i]));
    y1y16[i][1] = __halves2half2(__float2half(scales[i]), __float2half(scales[i]));
  }
  
  // 列结果
  float block_c[m_count][4] = {};
  
  // 反量化和乘法
  int k = offset_k;
  while (k < end_k) {
    if (k == nextgroup) {
      group++;
      nextgroup += groupsize;
      // 重新加载组参数
      for (int i = 0; i < 4; i++) {
        zeros[i] = b_gptq_qzeros[group * (size_n / 8) + (n + i) / 8];
        scales[i] = __half2float(b_gptq_scales[group * size_n + n + i]);
      }
    }
    
#pragma unroll
    for (int j = 0; j < 4; j++) {
      const int4* b_ptr4 = (int4*)b_ptr;
      int4 load_int4 = *b_ptr4;
      
      half2 dq[4][4];
      dequant_4bit_8_gptq(load_int4.x, dq[0], z1z16[0], y1y16[0], size_n, false);
      dequant_4bit_8_gptq(load_int4.y, dq[1], z1z16[1], y1y16[1], size_n, false);
      dequant_4bit_8_gptq(load_int4.z, dq[2], z1z16[2], y1y16[2], size_n, false);
      dequant_4bit_8_gptq(load_int4.w, dq[3], z1z16[3], y1y16[3], size_n, false);
      
#pragma unroll
      for (int m = 0; m < m_count; m++) {
        block_c[m][0] = fma(dot22_8_f(dq[0], a_ptr + m * a_stride), scales[0], block_c[m][0]);
        block_c[m][1] = fma(dot22_8_f(dq[1], a_ptr + m * a_stride), scales[1], block_c[m][1]);
        block_c[m][2] = fma(dot22_8_f(dq[2], a_ptr + m * a_stride), scales[2], block_c[m][2]);
        block_c[m][3] = fma(dot22_8_f(dq[3], a_ptr + m * a_stride), scales[3], block_c[m][3]);
      }
      
      b_ptr += size_n;
      a_ptr += 8;
    }
    
    k += 32;
  }
  
  // 存储结果
  for (int m = 0; m < m_count; m++) {
    half2* out = (half2*)&c[(offset_m + m) * size_n + n];
    half2 result01 = __halves2half2(__float2half_rn(block_c[m][0]), __float2half_rn(block_c[m][1]));
    half2 result23 = __halves2half2(__float2half_rn(block_c[m][2]), __float2half_rn(block_c[m][3]));
    atomicAdd(out, result01);
    atomicAdd(out + 1, result23);
  }
}

// 内核选择函数
typedef void (*fp_gemm_half_q_half_gptq_kernel)(const half*, const uint32_t*,
                                                const uint32_t*, const half*,
                                                half*, const int, const int,
                                                const int, const int);

template <bool first_block, int m_count>
__global__ void gemm_half_q_half_gptq_4bit_kernel_wrapper(
    const half* a, const uint32_t* b_q_weight,
    const uint32_t* b_gptq_qzeros,
    const half* b_gptq_scales, half* c,
    const int size_m, const int size_n, const int size_k, const int groups) {
  gemm_half_q_half_gptq_4bit_kernel<first_block, m_count><<<1, 1>>>(
      a, b_q_weight, b_gptq_qzeros, b_gptq_scales, c, size_m, size_n, size_k, groups);
}

fp_gemm_half_q_half_gptq_kernel pick_gemm_half_q_half_gptq_kernel(
    bool first_block, const int m_count) {
  if (m_count == 1) return gemm_half_q_half_gptq_4bit_kernel_wrapper<true, 1>;
  if (m_count == 2) return gemm_half_q_half_gptq_4bit_kernel_wrapper<true, 2>;
  if (m_count == 3) return gemm_half_q_half_gptq_4bit_kernel_wrapper<true, 3>;
  if (m_count == 4) return gemm_half_q_half_gptq_4bit_kernel_wrapper<true, 4>;
  if (m_count == 5) return gemm_half_q_half_gptq_4bit_kernel_wrapper<true, 5>;
  if (m_count == 6) return gemm_half_q_half_gptq_4bit_kernel_wrapper<true, 6>;
  if (m_count == 7) return gemm_half_q_half_gptq_4bit_kernel_wrapper<true, 7>;
  if (m_count == 8) return gemm_half_q_half_gptq_4bit_kernel_wrapper<true, 8>;
  return NULL;
}

void gemm_half_q_half_cuda_part(const half* a, const uint32_t* b_q_weight,
                                const uint32_t* b_gptq_qzeros,
                                const half* b_gptq_scales, half* c,
                                int size_m, int size_n, int size_k,
                                int m_count, int groups) {
  dim3 blockDim, gridDim;
  blockDim.x = BLOCK_KN_SIZE;
  blockDim.y = 1;
  blockDim.z = 1;
  gridDim.x = DIVIDE(size_n, BLOCK_KN_SIZE * 4);
  gridDim.y = DIVIDE(size_m, m_count);
  gridDim.z = DIVIDE(size_k, BLOCK_KN_SIZE);
  
  fp_gemm_half_q_half_gptq_kernel kernel =
      pick_gemm_half_q_half_gptq_kernel(true, m_count);
  
  const hipStream_t stream = at::cuda::getCurrentCUDAStream();
  kernel<<<gridDim, blockDim, 0, stream>>>(a, b_q_weight, b_gptq_qzeros,
                                           b_gptq_scales, c, size_m, size_n,
                                           size_k, groups);
}

// Python接口 - 适配你的代码格式
torch::Tensor fused_gptq_gemm_4bit_cuda(
    torch::Tensor input,
    torch::Tensor qweight,
    torch::Tensor qzeros,
    torch::Tensor scales,
    int groupsize
) {
  // 获取维度 - 适配你的格式
  int M = input.size(0);
  int K = input.size(1);
  int N = qweight.size(0);  // qweight=[N, K//8]
  int num_groups = scales.size(0);  // scales=[num_groups, K]
  
  // 创建输出张量
  auto output = torch::zeros({M, N}, torch::TensorOptions()
      .dtype(input.dtype())
      .device(input.device()));
  
  // 转换数据类型和格式以适配vLLM内核
  // 1. 转换qweight: [N, K//8] -> [K//8, N]
  auto qweight_transposed = qweight.t().contiguous();
  
  // 2. 转换qzeros: [num_groups, K//8] -> [num_groups, K//8] (保持不变)
  auto qzeros_contiguous = qzeros.contiguous();
  
  // 3. 转换scales: [num_groups, K] -> [num_groups, N] (需要重新排列)
  auto scales_reshaped = scales.contiguous();
  
  // 使用vLLM风格的分块策略
  int max_chunks = M / BLOCK_M_SIZE_MAX;
  int last_chunk = max_chunks * BLOCK_M_SIZE_MAX;
  int last_chunk_size = M - last_chunk;
  
  if (max_chunks) {
    gemm_half_q_half_cuda_part(
        input.data_ptr<at::Half>(),
        qweight_transposed.data_ptr<uint32_t>(),
        qzeros_contiguous.data_ptr<uint32_t>(),
        scales_reshaped.data_ptr<at::Half>(),
        output.data_ptr<at::Half>(),
        last_chunk, N, K, BLOCK_M_SIZE_MAX, num_groups
    );
  }
  
  if (last_chunk_size) {
    gemm_half_q_half_cuda_part(
        input.data_ptr<at::Half>() + last_chunk * K,
        qweight_transposed.data_ptr<uint32_t>(),
        qzeros_contiguous.data_ptr<uint32_t>(),
        scales_reshaped.data_ptr<at::Half>(),
        output.data_ptr<at::Half>() + last_chunk * N,
        last_chunk_size, N, K, last_chunk_size, num_groups
    );
  }
  
  hipDeviceSynchronize();
  return output;
}

// PyTorch绑定
PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("fused_gptq_gemm_4bit_cuda", &fused_gptq_gemm_4bit_cuda, "Fused GPTQ 4bit GEMM CUDA");
}
